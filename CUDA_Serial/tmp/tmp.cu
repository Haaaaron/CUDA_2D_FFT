
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <assert.h>

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}

/*********************/
/* CUFFT ERROR CHECK */
/*********************/
static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "CUFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "CUFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "CUFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "CUFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "CUFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "CUFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "CUFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "CUFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "CUFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "CUFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
                fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                           _cudaGetErrorEnum(err)); \
             hipDeviceReset(); assert(0); \
    }
}

/********/
/* MAIN */
/********/
int main() {

    hipfftHandle forward_plan, inverse_plan; 

    int batch = 3;
    int rank = 1;

    int nRows = 1;
    int nCols = 10;
    int n[1] = {nCols};

    int idist = nRows*nCols;
    int odist = nRows*(nCols/2+1);

    int inembed[] = {nCols};
    int onembed[] = {nCols/2+1};

    int istride = 1;
    int ostride = 1;

    cufftSafeCall(hipfftPlanMany(&forward_plan,  rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_D2Z, batch));

    hipfftDoubleReal *h_in = (hipfftDoubleReal*)malloc(sizeof(hipfftDoubleReal)*nRows*nCols*batch);
    for(int i=0; i<nRows*nCols*batch; i++) h_in[i] = 1.f;

    hipfftDoubleComplex* h_freq = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*nRows*(nCols/2+1)*batch);

    hipfftDoubleReal* d_in;            gpuErrchk(hipMalloc(&d_in, sizeof(hipfftDoubleReal)*nRows*nCols*batch)); 
    hipfftDoubleComplex* d_freq; gpuErrchk(hipMalloc(&d_freq, sizeof(hipfftDoubleComplex)*nRows*(nCols/2+1)*batch)); 

    gpuErrchk(hipMemcpy(d_in,h_in,sizeof(hipfftDoubleReal)*nRows*nCols*batch,hipMemcpyHostToDevice));

    cufftSafeCall(hipfftExecD2Z(forward_plan, d_in, d_freq));

    gpuErrchk(hipMemcpy(h_freq,d_freq,sizeof(hipfftDoubleComplex)*nRows*(nCols/2+1)*batch,hipMemcpyDeviceToHost));

    for(int i=0; i<nRows*(nCols/2+1)*batch; i++) printf("Direct transform: %i %f %f\n",i,h_freq[i].x,h_freq[i].y); 

    cufftSafeCall(hipfftPlanMany(&inverse_plan, rank, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2D, batch));

    cufftSafeCall(hipfftExecZ2D(inverse_plan, d_freq, d_in));

    gpuErrchk(hipMemcpy(h_in,d_in,sizeof(hipfftDoubleReal)*nRows*nCols*batch,hipMemcpyDeviceToHost));

    for(int i=0; i<nRows*nCols*batch; i++) printf("Inverse transform: %i %f \n",i,h_in[i]); 

    getchar();

}